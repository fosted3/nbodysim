#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <vector>
#include "hip/hip_vector_types.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <cassert>
#include "cuda_types.h"
#include <iostream>
#include "stdio.h"

#ifdef DOUBLE
#ifndef datatype3
#define datatype3 double3
#endif
#ifndef datatype
#define datatype double
#endif
#endif
#ifdef FLOAT
#ifndef datatype3
#define datatype3 float3
#endif
#ifndef datatype
#define datatype float
#endif
#endif
#ifndef datatype3
#error "DOUBLE / FLOAT undefined, use -DDOUBLE or -DFLOAT"
#endif

__device__ datatype length(datatype3 v)
{
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ void mul_datatype3(datatype3 &a, const datatype &b)
{
	a.x *= b;
	a.y *= b;
	a.z *= b;
}

__global__ void compute(cparticle *particles, unsigned int p_size, cnode *nodes, unsigned int n_size, datatype3 *results)
{
	uint16_t tid = threadIdx.x;
	uint32_t bid = blockIdx.x;
	__shared__ datatype3 acc[shared_size];
	__shared__ cnode dep[shared_size];
	__shared__ cparticle par;
	__shared__ datatype r_sq[shared_size];
	if (tid == 0)
	{
		par = particles[bid];
	}
	__syncthreads();
	if (tid < par.size)
	{
		dep[tid] = nodes[particles[bid].dependants[tid]];
		acc[tid].x = dep[tid].pos.x - par.pos.x;
		acc[tid].y = dep[tid].pos.y - par.pos.y;
		acc[tid].z = dep[tid].pos.z - par.pos.z;
		r_sq[tid] = pow(length(acc[tid]), -2.0f);
		r_sq[tid] *= 6.67384e-11f * dep[tid].mass / length(acc[tid]);
		mul_datatype3(acc[tid], r_sq[tid]);
	}
	__syncthreads();	
	for (unsigned int s = 1; s < shared_size; s *= 2)
	{
		if (tid % (2 * s) == 0 && tid + s < par.size)
		{
			acc[tid].x = acc[tid].x + acc[tid + s].x;
			acc[tid].y = acc[tid].y + acc[tid + s].y;
			acc[tid].z = acc[tid].z + acc[tid + s].z;
		}
		__syncthreads();
	}
	results[bid] = acc[0];
}

void run_compute(std::vector<cparticle> *host_particles_vector, std::vector<cnode> *host_nodes_vector, std::vector<datatype3> *host_results_vector)
{
	thrust::device_vector<cparticle> device_particles(host_particles_vector -> begin(), host_particles_vector -> end());
	thrust::device_vector<cnode> device_nodes(host_nodes_vector -> begin(), host_nodes_vector -> end());
	thrust::device_vector<datatype3> device_results(host_results_vector -> begin(), host_results_vector -> end());
	dim3 grid(device_particles.size(),1,1);
	dim3 block(shared_size,1,1);
	cparticle* device_particles_ptr = thrust::raw_pointer_cast(&device_particles[0]);
	cnode* device_nodes_ptr = thrust::raw_pointer_cast(&device_nodes[0]);
	datatype3* device_results_ptr = thrust::raw_pointer_cast(&device_results[0]);
	assert(device_particles.size() == device_results.size());
	compute<<<grid, block>>>(device_particles_ptr, device_particles.size(), device_nodes_ptr, device_nodes.size(), device_results_ptr);
	assert(hipDeviceSynchronize() == hipSuccess);
	host_results_vector -> resize(device_results.size());
	thrust::copy(device_results.begin(), device_results.end(), host_results_vector -> begin());
	device_particles.clear();
	device_nodes.clear();
	device_results.clear();	
}
