#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <vector>
#include "hip/hip_vector_types.h"
#include <cassert>
#include "cuda_types.h"
#include <iostream>
#include "stdio.h"

#ifdef DOUBLE
#ifndef datatype3
#define datatype3 double3
#endif
#ifndef datatype
#define datatype double
#endif
#endif
#ifdef FLOAT
#ifndef datatype3
#define datatype3 float3
#endif
#ifndef datatype
#define datatype float
#endif
#endif
#ifndef datatype3
#error "DOUBLE / FLOAT undefined, use -DDOUBLE or -DFLOAT"
#endif

#define handle_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ datatype length(datatype3 v)
{
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ void mul_datatype3(datatype3 &a, const datatype &b)
{
	a.x *= b;
	a.y *= b;
	a.z *= b;
}

__global__ void compute(cparticle *particle, cnode *nodes, datatype3 *results)
{
	uint16_t tid = threadIdx.x;
	uint16_t bid = blockIdx.x;
	__shared__ datatype3 acc[shared_size];
	__shared__ cnode dep[shared_size];
	__shared__ cparticle par;
	__shared__ datatype r_sq[shared_size];
	if (tid == 0) { par = particle[bid]; }
	__syncthreads();
	if (tid < par.size)
	{
		dep[tid] = nodes[par.dependants[tid]];
		acc[tid].x = dep[tid].pos.x - par.pos.x;
		acc[tid].y = dep[tid].pos.y - par.pos.y;
		acc[tid].z = dep[tid].pos.z - par.pos.z;
		r_sq[tid] = pow(length(acc[tid]), -2.0f);
		r_sq[tid] *= 6.67384e-11f * dep[tid].mass / length(acc[tid]);
		mul_datatype3(acc[tid], r_sq[tid]);
	}
	__syncthreads();
	for (uint16_t s = 1; s < shared_size; s *= 2)
	{
		if (tid % (2 * s) == 0 && tid + s < par.size)
		{
			acc[tid].x = acc[tid].x + acc[tid + s].x;
			acc[tid].y = acc[tid].y + acc[tid + s].y;
			acc[tid].z = acc[tid].z + acc[tid + s].z;
		}
		__syncthreads();
	}
	results[bid] = acc[0];
}

void copy_to_gpu(cnode *cache_addr, uint32_t loc, hipStream_t *stream, cnode *data)
{	
	handle_error(hipMemcpy(&cache_addr[loc], data, sizeof(cnode), hipMemcpyHostToDevice));
}

cnode* init_cache(void)
{
	cnode *addr = NULL;
	handle_error(hipMalloc(&addr, cache_size * sizeof(cnode)));
	return addr;
}

void free_cache(cnode *addr)
{
	handle_error(hipFree(addr));
}

void init_streams(hipStream_t *streams)
{
	for(unsigned int i = 0; i < compute_threads; i++)
	{
		handle_error(hipStreamCreate(&streams[i]));
	}
}

void free_streams(hipStream_t *streams)
{
	for (unsigned int i = 0; i < compute_threads; i++)
	{
		handle_error(hipStreamDestroy(streams[i]));
	}
}

cparticle* allocate_particles(void)
{
	cparticle *addr = NULL;
	handle_error(hipMalloc(&addr, sizeof(cparticle) * block_size));
	return addr;
}

datatype3* allocate_results(void)
{
	datatype3 *addr = NULL;
	handle_error(hipMalloc(&addr, sizeof(datatype3) * block_size));
	return addr;
}

void free_particles(cparticle *addr)
{
	handle_error(hipFree(addr));
}

void free_results(datatype3 *addr)
{
	handle_error(hipFree(addr));
}

void run_compute(cparticle *par, cparticle *par_addr, cnode *cache, hipStream_t *stream, datatype3 *results, datatype3 *res_addr, uint16_t size)
{
	//std::cout << "Running compute on " << 
	handle_error(hipMemcpyAsync(par_addr, par, sizeof(cparticle) * size, hipMemcpyHostToDevice, *stream));
	compute<<<size, shared_size, 0, *stream>>>(par_addr, cache, res_addr);
	hipStreamSynchronize(*stream);
	handle_error(hipMemcpyAsync(results, res_addr, sizeof(datatype3) * size, hipMemcpyDeviceToHost, *stream));
}

void call_dev_reset(void)
{
	hipDeviceReset();
}
