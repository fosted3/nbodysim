#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <vector>
#include "hip/hip_vector_types.h"
#include <cassert>
#include "cuda_types.h"
#include <iostream>
#include "stdio.h"

#ifdef DOUBLE
#ifndef datatype3
#define datatype3 double3
#endif
#ifndef datatype
#define datatype double
#endif
#endif
#ifdef FLOAT
#ifndef datatype3
#define datatype3 float3
#endif
#ifndef datatype
#define datatype float
#endif
#endif
#ifndef datatype3
#error "DOUBLE / FLOAT undefined, use -DDOUBLE or -DFLOAT"
#endif

#define handle_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ datatype length(datatype3 v)
{
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ void mul_datatype3(datatype3 &a, const datatype &b)
{
	a.x *= b;
	a.y *= b;
	a.z *= b;
}

__global__ void compute(cparticle *particle, cnode *nodes, datatype3 *results)
{
	uint16_t tid = threadIdx.x;
	uint16_t bid = blockIdx.x;
	__shared__ datatype3 acc[shared_size];
	__shared__ cnode dep[shared_size];
	__shared__ cparticle par;
	__shared__ datatype r_sq[shared_size];
	if (tid == 0) { par = particle[bid]; }
	__syncthreads();
	if (tid < par.size)
	{
		dep[tid] = nodes[par.dependants[tid]];
		acc[tid].x = dep[tid].pos.x - par.pos.x;
		acc[tid].y = dep[tid].pos.y - par.pos.y;
		acc[tid].z = dep[tid].pos.z - par.pos.z;
		r_sq[tid] = pow(length(acc[tid]), -2.0f);
		r_sq[tid] *= 6.67384e-11f * dep[tid].mass / length(acc[tid]);
		mul_datatype3(acc[tid], r_sq[tid]);
	}
	__syncthreads();
	for (uint16_t s = 1; s < shared_size; s *= 2)
	{
		if (tid % (2 * s) == 0 && tid + s < par.size)
		{
			acc[tid].x = acc[tid].x + acc[tid + s].x;
			acc[tid].y = acc[tid].y + acc[tid + s].y;
			acc[tid].z = acc[tid].z + acc[tid + s].z;
		}
		__syncthreads();
	}
	results[bid] = acc[0];
}

void init_streams(hipStream_t *streams)
{
	for(unsigned int i = 0; i < compute_threads; i++)
	{
		//std::cout << "Initializing stream " << i << std::endl;
		handle_error(hipStreamCreate(&(streams[i])));
	}
}

void free_streams(hipStream_t *streams)
{
	for (unsigned int i = 0; i < compute_threads; i++)
	{
		//std::cout << "Destroying stream " << i << std::endl;
		handle_error(hipStreamDestroy(streams[i]));
	}
}

cparticle* allocate_particles(void)
{
	cparticle *addr = NULL;
	handle_error(hipMalloc(&addr, sizeof(cparticle) * block_size));
	return addr;
}

cnode* allocate_nodes(void)
{
	cnode *addr = NULL;
	handle_error(hipMalloc(&addr, sizeof(cnode) * block_size * shared_size));
	return addr;
}

datatype3* allocate_results(void)
{
	datatype3 *addr = NULL;
	handle_error(hipMalloc(&addr, sizeof(datatype3) * block_size));
	return addr;
}

void free_particles(cparticle *addr)
{
	handle_error(hipFree(addr));
}

void free_nodes(cnode *addr)
{
	handle_error(hipFree(addr));
}

void free_results(datatype3 *addr)
{
	handle_error(hipFree(addr));
}

void run_compute(cparticle *particles, cparticle *par_addr, cnode *node, cnode *node_addr, datatype3 *results, datatype3 *res_addr, uint32_t par_size, uint32_t node_size, uint16_t threads, hipStream_t *stream)
{
	handle_error(hipMemcpyAsync(par_addr, particles, sizeof(cparticle) * par_size, hipMemcpyHostToDevice, *stream));
	handle_error(hipMemcpyAsync(node_addr, node, sizeof(cnode) * node_size, hipMemcpyHostToDevice, *stream));
	compute<<<par_size, threads, 0, *stream>>>(par_addr, node_addr, res_addr);
	handle_error(hipMemcpyAsync(results, res_addr, sizeof(datatype3) * par_size, hipMemcpyDeviceToHost, *stream));
	handle_error(hipStreamSynchronize(*stream));
}

void call_dev_reset(void)
{
	hipDeviceReset();
}
